
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void convert_nv12_bt701_block_linear(uint8_t *out, uint8_t *luma, uint8_t *chroma, uint32_t width, uint32_t height) {

    //TODO these need to be passed in
    uint32_t gobWidth    = 16;//px TODO calculate these from hardware
    uint32_t gobHeight   = 8;//px
    uint32_t log2GobsPerBlockX = 2;
    uint32_t log2GobsPerBlockY = 4;
    uint32_t bytesPerPixel = 4;//bpc * channels / 8;

    uint32_t blockWidth  = gobWidth * (1<<log2GobsPerBlockX);//px
    uint32_t blockHeight = gobHeight * (1<<log2GobsPerBlockY);//px

    uint32_t gobSize     = gobWidth * gobHeight * bytesPerPixel;
    uint32_t gobsPerBlockY = blockHeight / gobHeight;

    uint32_t blocksPerX  = (width+blockWidth-1)/blockWidth;
    uint32_t blockSize   = blockWidth * blockHeight * bytesPerPixel;

    uint32_t blockX      = blockIdx.x;
    uint32_t blockY      = blockIdx.y;
    uint32_t gobX        = threadIdx.x;
    uint32_t gobY        = threadIdx.y;

    uint32_t blockOffset = ((blockY * blocksPerX) + blockX) * blockSize;
    uint32_t gobOffset   = ((gobX * gobsPerBlockY) + gobY) * gobSize; //x and y are flipped for GOBs in blocks

    uint32_t gobPixelX   = blockX * blockWidth + gobX * gobWidth;
    uint32_t gobPixelY   = blockY * blockHeight + gobY * gobHeight;

    uint32_t subGobWidth  = 4;//px
    uint32_t subGobHeight = 4;//px

    for (uint32_t i = 0; i < gobSize; i+=4) {
        uint32_t t = i / bytesPerPixel;
        uint32_t idx = (i / 64);
        uint32_t half = idx / 4;

        uint32_t subGobX = idx&1;
        uint32_t subGobY = (idx&2)>>1;

        uint32_t subSubGobX = t&3;
        uint32_t subSubGobY = (t%16)/4;

        uint32_t x = gobPixelX + (half * subGobWidth * 2) + (subGobX * subGobWidth) + subSubGobX;
        uint32_t y = gobPixelY + (subGobY * subGobHeight) + subSubGobY;

        uint32_t pixelOffset = i;
        uint8_t *pixelOut =  out + blockOffset + gobOffset + pixelOffset;

//        pixelOut[2] = t;//R
//        pixelOut[1] = idx;//G
//        pixelOut[0] = 0;//B

        uint8_t *lumaOffset   = luma   + (y*width) + x;
        uint8_t *chromaOffset = chroma + (y>>1)*width + (x & ~1);

        uint8_t Y = lumaOffset[0];
        uint8_t U = chromaOffset[0];
        uint8_t V = chromaOffset[1];

        pixelOut[2] = (uint8_t) fmaxf(fminf(Y + 1.402 * (V - 128), 255.0), 0.0);
        pixelOut[1] = (uint8_t) fmaxf(fminf(Y - 0.34413 * (U - 128) - 0.71414*(V - 128), 255.0), 0.0);
        pixelOut[0] = (uint8_t) fmaxf(fminf(Y + 1.772*(U - 128), 255.0), 0.0);
    }
}
